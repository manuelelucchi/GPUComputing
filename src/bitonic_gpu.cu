
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#define THREADS 256
#define BLOCKS 8 * 1024

#define CHECK(call)                                                \
    {                                                              \
        const hipError_t error = call;                            \
        if (error != hipSuccess)                                  \
        {                                                          \
            fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
            fprintf(stderr, "code: %d, reason: %s\n", error,       \
                    hipGetErrorString(error));                    \
        }                                                          \
    }

__global__ void bitonic_sort_step(int *a, int j, int k)
{
    unsigned int i, ixj;
    i = threadIdx.x + blockDim.x * blockIdx.x;
    ixj = i ^ j;

    if ((ixj) > i)
    {
        if ((i & k) == 0)
        {
            if (a[i] > a[ixj])
            {
                int temp = a[i];
                a[i] = a[ixj];
                a[ixj] = temp;
            }
        }
        if ((i & k) != 0)
        {
            if (a[i] < a[ixj])
            {
                int temp = a[i];
                a[i] = a[ixj];
                a[ixj] = temp;
            }
        }
    }
}

int main(void)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int N = THREADS * BLOCKS;
    // check
    if (!(N && !(N & (N - 1))))
    {
        printf("ERROR: N must be power of 2 (N = %d)\n", N);
        exit(1);
    }
    size_t nBytes = N * sizeof(int);
    int *a = (int *)malloc(nBytes);
    int *b = (int *)malloc(nBytes);

    // fill data
    for (int i = 0; i < N; ++i)
    {
        a[i] = rand() % 100;
        b[i] = a[i];
    }

    // device mem copy
    int *d_a;
    CHECK(hipMalloc((void **)&d_a, nBytes));
    CHECK(hipMemcpy(d_a, a, nBytes, hipMemcpyHostToDevice));

    // num of threads
    dim3 blocks(BLOCKS, 1);   // Number of blocks
    dim3 threads(THREADS, 1); // Number of threads

    // start computation
    hipEventRecord(start);
    int j, k;
    // external loop on comparators of size k
    for (k = 2; k <= N; k <<= 1)
    {
        // internal loop for comparator internal stages
        for (j = k >> 1; j > 0; j = j >> 1)
            bitonic_sort_step<<<blocks, threads>>>(d_a, j, k);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU elapsed time: %.5f (sec)\n", milliseconds / 1000);

    // recover data
    hipMemcpy(a, d_a, nBytes, hipMemcpyDeviceToHost);

    hipFree(d_a);
    exit(0);
}